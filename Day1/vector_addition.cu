
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#define ll long long int

__global__ // Marks a function as a kernel function that runs on the GPU. Callable from the Host (CPU) code. Runs on the Device (GPU).
// __device__ // Marks a function as a device function that runs on the GPU. Callable from other device or global functions. Runs on the Device (GPU).
void vecAddKernel(float* A, float* B, float* C, ll n){
	ll i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i < n)
		C[i] = B[i] + A[i];
}
void vecAdd(float* A, float* B, float* C, ll n){
	float *A_d, *B_d, *C_d;
	ll size = n * sizeof(float);
	hipMalloc((void **) &A_d, size);
	hipMalloc((void **) &B_d, size);
	hipMalloc((void **) &C_d, size);
	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
	vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);
	hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
}

void linearVecAdd(float* A, float* B, float* C, ll n){
    for(ll i = 0; i < n; i++)
        C[i] = A[i] + B[i];
}
int main()
{
    ll n = 1e4;
    float* A = new float[n];
    float* B = new float[n];
    float* C = new float[n];
    for(ll i = 0; i < n; i++) {
        A[i] = static_cast<float>(i);
        B[i] = static_cast<float>(i * 2);
    }
    // compare the time between vecAdd and linearVecAdd
    float timeVecAdd, timeLinearVecAdd;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vecAdd(A, B, C, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeVecAdd, start, stop);
    cout << "Time taken by vecAdd: " << timeVecAdd << " ms" << endl;

    hipEventRecord(start);
    linearVecAdd(A, B, C, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeLinearVecAdd, start, stop);
    cout << "Time taken by linearVecAdd: " << timeLinearVecAdd << " ms" << endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}

// Execution:
// nvcc vector_addition.cu -o vector_addition
// nvprof ./vector_addition
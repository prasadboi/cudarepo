
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
using namespace std;
#define TILE_WIDTH 16
template<typename T>
__global__ void matrixMulKernelTiled(T const* M, T const* N, T* P, int h1, int w1, int h2, int w2)
{
	__shared__ T M_d_shared[TILE_WIDTH][TILE_WIDTH];
	__shared__ T N_d_shared[TILE_WIDTH][TILE_WIDTH];

	int row = blockIdx.y*TILE_WIDTH + threadIdx.y;
	int col = blockIdx.x*TILE_WIDTH + threadIdx.x;

	// assume h1 == w2
	T value = 0;
    int numTiles = (w1 + TILE_WIDTH - 1)/TILE_WIDTH;
	for(size_t ph = 0; ph < numTiles; ++ph)
	{
		// load the M and N tiles into shared memory
		if ((row < h1) and ((ph*TILE_WIDTH + threadIdx.x) < w1))
			M_d_shared[threadIdx.y][threadIdx.x] = M[row*w1 + ph*TILE_WIDTH + threadIdx.x];
		else
			M_d_shared[threadIdx.y][threadIdx.x] = 0.0f;
			
		if (((ph*TILE_WIDTH + threadIdx.y) < h2) and (col < w2))
			N_d_shared[threadIdx.y][threadIdx.x] = N[(ph*TILE_WIDTH + threadIdx.y)*w2 + col];
		else
			N_d_shared[threadIdx.y][threadIdx.x] = 0.0f;
			
		__syncthreads();
		for(size_t k = 0; k < TILE_WIDTH; k++)
		{
			value += (M_d_shared[threadIdx.y][k]*N_d_shared[k][threadIdx.x]);
		}
		__syncthreads();
	}
	if((row < h1) and (col < w2))
		P[row*w2 + col] = value;
}

template<typename T>
__host__
void launch_GMEM_kernel(int kernel_type, T const* M, T const* N, T* P, size_t h1, size_t w1, size_t h2, size_t w2)
{
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((w2 + TILE_WIDTH - 1) / TILE_WIDTH, (h1 + TILE_WIDTH - 1) / TILE_WIDTH) ;
    switch (kernel_type)
    {
    case 1:
        matrixMulKernelTiled<T><<<gridDim, blockDim>>>(M, N, P, h1, w1, h2, w2);
        break;
    
    default:
        cout<<"unidentified kernel launch type. No execution.\n";
        break;
    }
}

__host__
void MatrixMul(float* M, float* N, float* P, int h1, int w1, int h2, int w2)
{
    // Assume w1 == h2
    for (int row = 0; row < h1; ++row) {
        for (int col = 0; col < w2; ++col) {
            float value = 0.0f;
            for (int k = 0; k < w1; ++k) {
                value += M[row * w1 + k] * N[k * w2 + col];
            }
            P[row * w2 + col] = value;
        }
    }
}
int main() {
    // Matrix dimensions
    const int h1 = 16;
    const int w1 = 32;
    const int h2 = 32;
    const int w2 = 16;

    if (w1 != h2) {
        std::cerr << "Matrix dimensions invalid for multiplication!" << std::endl;
        return 1;
    }

    float* M_h = new float[h1 * w1];
    float* N_h = new float[h2 * w2];
    float* P_h = new float[h1 * w2];
    float* P_test_h = new float[h1 * w2];

    for (int i = 1; i <= h1 * w1; ++i) {
        M_h[i-1] = 1.0f*i;
    }
    for (int i = 1; i <= h2 * w2; ++i) {
        N_h[i-1] = 1.0f*i;
    }

    // Allocate device memory
    float *M_d, *N_d, *P_d;
    hipMalloc(&M_d, h1 * w1 * sizeof(float));
    hipMalloc(&N_d, h2 * w2 * sizeof(float));
    hipMalloc(&P_d, h1 * w2 * sizeof(float));

    // Copy data to device
    hipMemcpy(M_d, M_h, h1 * w1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, h2 * w2 * sizeof(float), hipMemcpyHostToDevice);

    launch_GMEM_kernel<float>(1, M_d, N_d, P_d, h1, w1, h2, w2);
    hipDeviceSynchronize();

    hipMemcpy(P_h, P_d, h1 * w2 * sizeof(float), hipMemcpyDeviceToHost);

    // checking if the output of the GPU kernel matches the actual expected output
    // getting accurate results from the naive, single thread matrix multiplication program
    MatrixMul(M_h, N_h, P_test_h, h1, w1, h2, w2);
    // Compare outputs only on the host (driver) thread
    bool correct = true;
    for(size_t i = 0; i < h1; i++)
    {
        for(size_t j = 0; j < w2; j++)
        {
            if(fabs(P_h[i*w2 + j] - P_test_h[i*w2 + j]) > 1e-4)
            {
                cout << "Mismatch at (" << i << "," << j << "): "
                     << "GPU=" << P_h[i*w2 + j] << ", CPU=" << P_test_h[i*w2 + j] << endl;
                correct = false;
            }
        }
    }
    if (correct)
        cout << "GPU Kernel output matches CPU output." << endl;
    else
        cout << "GPU Kernel provides wrong output!!!" << endl;
    // Cleanup
    delete[] M_h;
    delete[] N_h;
    delete[] P_h;
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);

    return 0;
}
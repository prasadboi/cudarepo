
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

// P[row, col] = sum(M[row,k] * N[k, col]) for k = 0,1,2...,width-1
__global__
void MatrixMulKernel(float* M, float* N, float* P, int h1, int w1, int h2, int w2){
	// note that w1 must be equal to w2
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if((row < h1) and (col < w2)){
		float value = 0.0;
		for(int k = 0; k < w1; k++){
			value += (M[row*w1 + k] * N[k*w2 + col]); // note the way the row major format is being applied to fetch the matrix values
		}
		P[row*w2 + col] = value;
	}
}

__host__
void MatrixMul(float* M, float* N, float* P, int h1, int w1, int h2, int w2)
{
    // Assume w1 == h2
    for (int row = 0; row < h1; ++row) {
        for (int col = 0; col < w2; ++col) {
            float value = 0.0f;
            for (int k = 0; k < w1; ++k) {
                value += M[row * w1 + k] * N[k * w2 + col];
            }
            P[row * w2 + col] = value;
        }
    }
}
int main() {
    // Matrix dimensions
    const int h1 = 128;
    const int w1 = 256;
    const int h2 = 256;
    const int w2 = 64;

    if (w1 != h2) {
        std::cerr << "Matrix dimensions invalid for multiplication!" << std::endl;
        return 1;
    }

    float* M_h = new float[h1 * w1];
    float* N_h = new float[h2 * w2];
    float* P_h = new float[h1 * w2];
    float* P_test_h = new float[h1 * w2];

    for (int i = 1; i <= h1 * w1; ++i) {
        M_h[i-1] = 1.0f*i;
    }
    for (int i = 1; i <= h2 * w2; ++i) {
        N_h[i-1] = 1.0f*i;
    }

    // Allocate device memory
    float *M_d, *N_d, *P_d;
    hipMalloc(&M_d, h1 * w1 * sizeof(float));
    hipMalloc(&N_d, h2 * w2 * sizeof(float));
    hipMalloc(&P_d, h1 * w2 * sizeof(float));

    // Copy data to device
    hipMemcpy(M_d, M_h, h1 * w1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, h2 * w2 * sizeof(float), hipMemcpyHostToDevice);

    // Kernel launch parameters
    dim3 blockDim(25, 25);
    // note how the grid dims are being calculated!!!
    dim3 gridDim((h1 + blockDim.x - 1) / blockDim.x, (w2 + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    MatrixMulKernel<<<gridDim, blockDim>>>(M_d, N_d, P_d, h1, w1, h2, w2);
    hipDeviceSynchronize();

    hipMemcpy(P_h, P_d, h1 * w2 * sizeof(float), hipMemcpyDeviceToHost);

    // Print a small part of the result
    cout << "Kernel function output (first 5x5 block):" << endl;
    for (int i = 0; i < 5 && i < h1; ++i) {
        for (int j = 0; j < 5 && j < w2; ++j) {
            cout << P_h[i * w2 + j] << " ";
        }
        cout << endl;
    }
    cout<< "------------------------" << endl;
    MatrixMul(M_h, N_h, P_test_h, h1, w1, h2, w2); // CPU version for testing
    cout << "Ideal output (first 5x5 block):" << endl;
    for (int i = 0; i < 5 && i < h1; ++i) {
        for (int j = 0; j < 5 && j < w2; ++j) {
            cout << P_test_h[i * w2 + j] << " ";
        }
        cout << endl;
    }
    cout<< "------------------------" << endl;
    // Cleanup
    delete[] M_h;
    delete[] N_h;
    delete[] P_h;
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);

    return 0;
}